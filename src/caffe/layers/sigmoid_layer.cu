#include "hip/hip_runtime.h"
// Copyright 2014 Tobias Domhan

#include "caffe/layer.hpp"
#include "caffe/vision_layers.hpp"
#include <algorithm>
#include <cmath>

// Kaixiang Mo
#include <iostream>
// ~Kaixiang Mo

using std::max;

namespace caffe {

template <typename Dtype>
inline Dtype sigmoid(Dtype x) {
  return 1. / (1. + exp(-x));
}

template <typename Dtype>
void SigmoidLayer<Dtype>::Forward_cpu(const vector<Blob<Dtype>*>& bottom,
    vector<Blob<Dtype>*>* top) {
  const Dtype* bottom_data = bottom[0]->cpu_data();
  Dtype* top_data = (*top)[0]->mutable_cpu_data();
  const int count = bottom[0]->count();
  for (int i = 0; i < count; ++i) {
    top_data[i] = sigmoid(bottom_data[i]);
  }
}

template <typename Dtype>
Dtype SigmoidLayer<Dtype>::Backward_cpu(const vector<Blob<Dtype>*>& top,
    const bool propagate_down,
    vector<Blob<Dtype>*>* bottom) {
  if (propagate_down) {
    const Dtype* bottom_data = (*bottom)[0]->cpu_data();
    const Dtype* top_diff = top[0]->cpu_diff();
    Dtype* bottom_diff = (*bottom)[0]->mutable_cpu_diff();
    const int count = (*bottom)[0]->count();
    for (int i = 0; i < count; ++i) {
      Dtype sigmoid_x = sigmoid(bottom_data[i]);
      bottom_diff[i] = top_diff[i] * sigmoid_x * (1. - sigmoid_x);
    }
  }
  return Dtype(0);
}


template <typename Dtype>
__device__ inline Dtype sigmoid_gpu(Dtype x) {
  return 1. / (1. + exp(-x));
}


template <typename Dtype>
__global__ void SigmoidForward(const int n, const Dtype* in, Dtype* out) {
  int index = threadIdx.x + blockIdx.x * blockDim.x;
  if (index < n) {
    out[index] = sigmoid_gpu(in[index]);
  }
}

template <typename Dtype>
void SigmoidLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
    vector<Blob<Dtype>*>* top) {
	
  // int num = bottom[0]->num();
  // int dim = bottom[0]->count() / num;
  // const Dtype* bottom_data1 = bottom[0]->cpu_data();
  // std::cout << "SigmoidLayer input" << std::endl;
  // for (int i = 0; i < 5; ++i) {
	// for (int j = 0; j < std::min(10,dim); ++j ){
	  // std::cout << bottom_data1[i * dim + j] << "\t";
	// }
	// std::cout << std::endl; 
  // }

  const Dtype* bottom_data = bottom[0]->gpu_data();
  Dtype* top_data = (*top)[0]->mutable_gpu_data();
  const int count = bottom[0]->count();
  
  SigmoidForward<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
      count, bottom_data, top_data);
  CUDA_POST_KERNEL_CHECK;
  // << " count: " << count << " bottom_data: "
  //     << (unsigned long)bottom_data << " top_data: " << (unsigned long)top_data
  //     << " blocks: " << CAFFE_GET_BLOCKS(count)
  //     << " threads: " << CAFFE_CUDA_NUM_THREADS;
  
  // const Dtype* top_data1 = (*top)[0]->cpu_data();
  // std::cout << "SigmoidLayer output" << std::endl;
  // for (int i = 0; i < 5; ++i) {
	// for (int j = 0; j < std::min(10,dim); ++j ){
	  // std::cout << top_data1[i * dim + j] << "\t";
	// }
	// std::cout << std::endl; 
  // }
  
}

template <typename Dtype>
__global__ void SigmoidBackward(const int n, const Dtype* in_diff,
    const Dtype* in_data, Dtype* out_diff) {
  int index = threadIdx.x + blockIdx.x * blockDim.x;
  if (index < n) {
    Dtype sigmoid_x = sigmoid_gpu(in_data[index]);
    out_diff[index] = in_diff[index] * sigmoid_x * (1 - sigmoid_x);
  }
}

template <typename Dtype>
Dtype SigmoidLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
    const bool propagate_down,
    vector<Blob<Dtype>*>* bottom) {
  if (propagate_down) {
    const Dtype* bottom_data = (*bottom)[0]->gpu_data();
    const Dtype* top_diff = top[0]->gpu_diff();
    Dtype* bottom_diff = (*bottom)[0]->mutable_gpu_diff();
    const int count = (*bottom)[0]->count();
    SigmoidBackward<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
        count, top_diff, bottom_data, bottom_diff);
    CUDA_POST_KERNEL_CHECK;
  }
  return Dtype(0);
}

INSTANTIATE_CLASS(SigmoidLayer);


}  // namespace caffe
